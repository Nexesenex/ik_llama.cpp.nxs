#include "hip/hip_runtime.h"
#include "../mmq_id_common.cuh"

template <int mmq_y, bool need_check> static __device__ __forceinline__ void load_tiles_iq6_k(
    const char * __restrict__ x, int * __restrict__ x_tile, const int kbx0, const int i_max, const int stride) {

    constexpr int nwarps = mmq_get_nwarps_device();

#ifdef INT8_MMA_AVAILABLE
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + WARP_SIZE*2);
#else
    constexpr tile_x_sizes txs = MMQ_DP4A_TXS_Q8_0_16;
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + txs.qs);
#endif // INT8_MMA_AVAILABLE

    constexpr int qstep = 8;
    const int kqsx = threadIdx.x % qstep;

    auto values = iq6nl_values;
    int qh[2];

    uint32_t aux32[2];
    const uint8_t * aux8 = (const uint8_t *)aux32;
#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * WARP_SIZE/qstep) {
        int i = i0 + threadIdx.y*(WARP_SIZE/qstep) + threadIdx.x/qstep;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_iq6_k * bxi = (const block_iq6_k *)(x + i*stride) + kbx0;

        const float d = bxi->d;
        uint16_t extra = bxi->extra >> (kqsx/4);

        qh[0] = get_int_b4(bxi->qh, kqsx+0);
        qh[1] = get_int_b4(bxi->qh, kqsx+8);

    #pragma unroll
        for (int l = 0; l < qstep/2; ++l) {

            const int ql = get_int_b4(bxi->qs, kqsx + qstep*l);
            aux32[0] = ((ql >> 0) & 0x0f0f0f0f) | ((qh[l/2] & 0x03030303) << 4) | ((extra & 1) * 0x40404040);
            aux32[1] = ((ql >> 4) & 0x0f0f0f0f) | ((qh[l/2] & 0x0c0c0c0c) << 2) | ((extra & 4) * 0x10101010);
            qh[l/2] >>= 4;
            extra   >>= 4;

            const char4 val0  = make_char4(values[aux8[0]], values[aux8[1]], values[aux8[2]], values[aux8[3]]);
            const char4 val1  = make_char4(values[aux8[4]], values[aux8[5]], values[aux8[6]], values[aux8[7]]);

#ifdef INT8_MMA_AVAILABLE
            x_qs[i*MMQ_MMA_TILE_X_K_Q3_K + kqsx + 16*l + 0] = *(const int *)&val0;
            x_qs[i*MMQ_MMA_TILE_X_K_Q3_K + kqsx + 16*l + 8] = *(const int *)&val1;
#else
            x_qs[i*(2*WARP_SIZE + 1)     + kqsx + 16*l + 0] = *(const int *)&val0;
            x_qs[i*(2*WARP_SIZE + 1)     + kqsx + 16*l + 8] = *(const int *)&val1;
#endif // INT8_MMA_AVAILABLE
        }


#ifdef INT8_MMA_AVAILABLE
        x_df[i*MMQ_MMA_TILE_X_K_Q3_K               + 2*kqsx+0] = d * bxi->scales[2*kqsx+0];
        x_df[i*MMQ_MMA_TILE_X_K_Q3_K               + 2*kqsx+1] = d * bxi->scales[2*kqsx+1];
#else
        x_df[i*(2*WARP_SIZE*2/QI8_0) + i/(QI8_0/4) + 2*kqsx+0] = d * bxi->scales[2*kqsx+0];
        x_df[i*(2*WARP_SIZE*2/QI8_0) + i/(QI8_0/4) + 2*kqsx+1] = d * bxi->scales[2*kqsx+1];
#endif // INT8_MMA_AVAILABLE
    }
}

template <int mmq_x, int mmq_y, bool need_check>
struct mmq_type_traits_id<mmq_x, mmq_y, need_check, GGML_TYPE_IQ6_K> {
    static constexpr load_tiles_mmq_t load_tiles   = load_tiles_iq6_k<mmq_y, need_check>;
    static constexpr vec_dot_mmq_t    vec_dot_mma  = vec_dot_q8_0_16_q8_1_mma<mmq_x, mmq_y>;
    static constexpr vec_dot_mmq_t    vec_dot_dp4a = vec_dot_q8_0_16_q8_1_dp4a<mmq_x, mmq_y>;
};

DECL_MMQ_CASE(GGML_TYPE_IQ6_K);
