#include "hip/hip_runtime.h"
#include "../mmq_id_common.cuh"

template <int mmq_y, bool need_check> static __device__ __forceinline__ void load_tiles_iq4_kss(
    const char * __restrict__ x, int * __restrict__ x_tile, const int kbx0, const int i_max, const int stride) {

    constexpr int nwarps = mmq_get_nwarps_device();

#ifdef INT8_MMA_AVAILABLE
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + WARP_SIZE*2);
#else
    constexpr tile_x_sizes txs = mmq_get_dp4a_tile_x_sizes(GGML_TYPE_IQ4_XS, mmq_y);
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + txs.qs);
#endif // INT8_MMA_AVAILABLE

    const int kqsx = threadIdx.x / 4;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += 4*nwarps) {
        int i = i0 + 4*threadIdx.y + threadIdx.x%4;

        if (need_check) {
            i = min(i, i_max);
        }

        const float * dptr = (const float *)(x + i*stride);
        const block_iq4_kss * bxi = (const block_iq4_kss *)(dptr + 1) + kbx0;
        const uint32_t * q4 = bxi->qs + 4*kqsx;
        uint32_t s32 = (q4[0] & 0x00010001) | ((q4[1] & 0x00010001) << 2) | ((q4[2] & 0x00010001) << 4) | ((q4[3] & 0x00010001) << 6);
        uint8_t ls = (s32 | (s32 >> 15)) & 0xff;

        auto values = iq4k_values + ((ls & 1) << 4);

        #pragma unroll
        for (int j = 0; j < 4; ++j) {
            uint32_t val = q4[j] & 0xfffefffe;
            val = val ^ (val >> 1);
            auto v = get_int_from_table_16(val, values);
#ifdef INT8_MMA_AVAILABLE
            x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + 8*kqsx + j + 0] = v.x;
            x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + 8*kqsx + j + 4] = v.y;
#else
            x_qs[i*(2*WARP_SIZE + 1)     + 8*kqsx + j + 0] = v.x;
            x_qs[i*(2*WARP_SIZE + 1)     + 8*kqsx + j + 4] = v.y;
#endif // INT8_MMA_AVAILABLE
        }
#ifdef INT8_MMA_AVAILABLE
        x_df[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx] = dptr[0] * ((ls & 254) - 127);
#else
        x_df[i*(WARP_SIZE/4) + i/4   + kqsx] = dptr[0] * ((ls & 254) - 127);
#endif // INT8_MMA_AVAILABLE
    }

}

template <int mmq_y, bool need_check> static __device__ __forceinline__ void load_tiles_iq4_ks(
    const char * __restrict__ x, int * __restrict__ x_tile, const int kbx0, const int i_max, const int stride) {

    constexpr int nwarps = mmq_get_nwarps_device();

#ifdef INT8_MMA_AVAILABLE
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + WARP_SIZE*2);
#else
    constexpr tile_x_sizes txs = mmq_get_dp4a_tile_x_sizes(GGML_TYPE_IQ4_XS, mmq_y);
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + txs.qs);
#endif // INT8_MMA_AVAILABLE

    const int kqsx = threadIdx.x / 4;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += 4*nwarps) {
        int i = i0 + 4*threadIdx.y + threadIdx.x%4;

        if (need_check) {
            i = min(i, i_max);
        }

        const float * dptr = (const float *)(x + i*stride);
        const block_iq4_ks * bxi = (const block_iq4_ks *)(dptr + 1) + kbx0;
        const int ls = (bxi->scales[kqsx] & 254) - 127;

        auto values = iq4k_values + ((bxi->scales[kqsx] & 1) << 4);

        #pragma unroll
        for (int j = 0; j < 4; ++j) {
            const int q4 = get_int_b4(bxi->qs, 4*kqsx+j);
            const int2 v = get_int_from_table_16(q4, values);
#ifdef INT8_MMA_AVAILABLE
            x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + 8*kqsx + j + 0] = v.x;
            x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + 8*kqsx + j + 4] = v.y;
#else
            x_qs[i*(2*WARP_SIZE + 1)     + 8*kqsx + j + 0] = v.x;
            x_qs[i*(2*WARP_SIZE + 1)     + 8*kqsx + j + 4] = v.y;
#endif // INT8_MMA_AVAILABLE
        }
#ifdef INT8_MMA_AVAILABLE
        x_df[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx] = dptr[0] * ls;
#else
        x_df[i*(WARP_SIZE/4) + i/4   + kqsx] = dptr[0] * ls;
#endif // INT8_MMA_AVAILABLE
    }

}

template <int mmq_y, bool need_check> static __device__ __forceinline__ void load_tiles_iq4_ks_r4(
    const char * __restrict__ x, int * __restrict__ x_tile, const int kbx0, const int i_max, const int stride) {

    constexpr int nwarps = mmq_get_nwarps_device();

#ifdef INT8_MMA_AVAILABLE
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + WARP_SIZE*2);
#else
    constexpr tile_x_sizes txs = mmq_get_dp4a_tile_x_sizes(GGML_TYPE_IQ4_KS_R4, mmq_y);
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + txs.qs);
#endif // INT8_MMA_AVAILABLE

    const int kqsx = threadIdx.x/4;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += 4*nwarps) {
        int i = i0 + 4*threadIdx.y + threadIdx.x%4;

        if (need_check) {
            i = min(i, i_max);
        }
        int i4 = i/4;
        int ir = i%4;

        const float * dptr = (const float *)(x + 4*i4*stride);
        const block_iq4_ks_r4 * bxi = (const block_iq4_ks_r4 *)(dptr + 4) + kbx0;

        const int ls = (bxi->scales[4*kqsx + ir] & 254) - 127;
        auto values = iq4k_values + ((bxi->scales[4*kqsx+ir] & 1) << 4);

#pragma unroll
        for (int j = 0; j < 4; ++j) {
            const int q4 = get_int_b4(bxi->qs, 16*kqsx+4*j+ir);
            const int2 v = get_int_from_table_16(q4, values);
            const int k0 = 8*kqsx + 4*(j%2) + j/2;
#ifdef INT8_MMA_AVAILABLE
            x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + k0 + 0] = v.x;
            x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + k0 + 2] = v.y;
#else
            x_qs[i*(2*WARP_SIZE + 1)     + k0 + 0] = v.x;
            x_qs[i*(2*WARP_SIZE + 1)     + k0 + 2] = v.y;
#endif // INT8_MMA_AVAILABLE
        }
#ifdef INT8_MMA_AVAILABLE
        x_df[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx] = dptr[ir] * ls;
#else
        x_df[i*(WARP_SIZE/4) + i/4   + kqsx] = dptr[ir] * ls;
#endif // INT8_MMA_AVAILABLE

    }

}

template <int mmq_x, int mmq_y, bool need_check>
struct mmq_type_traits_id<mmq_x, mmq_y, need_check, GGML_TYPE_IQ4_KSS> {
    static constexpr load_tiles_mmq_t load_tiles   = load_tiles_iq4_kss<mmq_y, need_check>;
    static constexpr vec_dot_mmq_t    vec_dot_mma  = vec_dot_q8_0_q8_1_mma<mmq_x, mmq_y, MMQ_Q8_1_DS_LAYOUT_D4>;
    static constexpr vec_dot_mmq_t    vec_dot_dp4a = vec_dot_q8_0_q8_1_dp4a<mmq_x, mmq_y>;
};

template <int mmq_x, int mmq_y, bool need_check>
struct mmq_type_traits_id<mmq_x, mmq_y, need_check, GGML_TYPE_IQ4_KS> {
    static constexpr load_tiles_mmq_t load_tiles   = load_tiles_iq4_ks<mmq_y, need_check>;
    static constexpr vec_dot_mmq_t    vec_dot_mma  = vec_dot_q8_0_q8_1_mma<mmq_x, mmq_y, MMQ_Q8_1_DS_LAYOUT_D4>;
    static constexpr vec_dot_mmq_t    vec_dot_dp4a = vec_dot_q8_0_q8_1_dp4a<mmq_x, mmq_y>;
};

template <int mmq_x, int mmq_y, bool need_check>
struct mmq_type_traits_id<mmq_x, mmq_y, need_check, GGML_TYPE_IQ4_KS_R4> {
    static constexpr load_tiles_mmq_t load_tiles   = load_tiles_iq4_ks_r4<mmq_y, need_check>;
    static constexpr vec_dot_mmq_t    vec_dot_mma  = vec_dot_q8_0_q8_1_mma<mmq_x, mmq_y, MMQ_Q8_1_DS_LAYOUT_D4>;
    static constexpr vec_dot_mmq_t    vec_dot_dp4a = vec_dot_q8_0_q8_1_dp4a<mmq_x, mmq_y>;
};

DECL_MMQ_CASE(GGML_TYPE_IQ4_KSS);
DECL_MMQ_CASE(GGML_TYPE_IQ4_KS);
DECL_MMQ_CASE(GGML_TYPE_IQ4_KS_R4);

