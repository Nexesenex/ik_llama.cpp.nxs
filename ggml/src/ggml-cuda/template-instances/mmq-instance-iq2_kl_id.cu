#include "hip/hip_runtime.h"
#include "../mmq_id_common.cuh"

template <int mmq_y, bool need_check> static __device__ __forceinline__ void load_tiles_iq2_kl(
    const char * __restrict__ x, int * __restrict__ x_tile, const int kbx0, const int i_max, const int stride) {

    constexpr int nwarps = mmq_get_nwarps_device();

#ifdef INT8_MMA_AVAILABLE
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + WARP_SIZE*2);
#else
    constexpr tile_x_sizes txs = mmq_get_dp4a_tile_x_sizes(GGML_TYPE_IQ4_XS, mmq_y);
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + txs.qs);
#endif // INT8_MMA_AVAILABLE

    const int kqsx = threadIdx.x/4;

    uint32_t aux32[2];
    const uint8_t * a8 = (const uint8_t *)aux32;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += 4*nwarps) {
        int i = i0 + 4*threadIdx.y + threadIdx.x%4;

        if (need_check) {
            i = min(i, i_max);
        }

        const half * dptr = (const half *)(x + i*stride);
        const float d = *dptr;
        const block_iq2_kl * bxi = (const block_iq2_kl *)(dptr + 1) + kbx0;

        #pragma unroll
        for (int j = 0; j < 2; ++j) {
            auto ql = get_int_b2(bxi->qs, 4*(kqsx/2) + 2*(kqsx%2) + j);
            auto qh = get_int_b2(bxi->qh, 2*(kqsx%2) + j) >> 2*(kqsx/2);
            aux32[0] = ((ql >> 0) & 0x0f0f0f0f) | ((qh << 4) & 0x10101010);
            aux32[1] = ((ql >> 4) & 0x0f0f0f0f) | ((qh << 3) & 0x10101010);
            #pragma unroll
            for (int l = 0; l < 2; ++l) {
                int val1 = iq2kl_values[a8[2*l+0]] | (iq2kl_values[a8[2*l+1]] << 16);
                int val2 = iq2kl_values[a8[2*l+4]] | (iq2kl_values[a8[2*l+5]] << 16);
#ifdef INT8_MMA_AVAILABLE
                x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + 16*(kqsx/2) + 4*(kqsx%2) + 2*j + l + 0] = val1;
                x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + 16*(kqsx/2) + 4*(kqsx%2) + 2*j + l + 8] = val2;
#else
                x_qs[i*(2*WARP_SIZE + 1)     + 16*(kqsx/2) + 4*(kqsx%2) + 2*j + l + 0] = val1;
                x_qs[i*(2*WARP_SIZE + 1)     + 16*(kqsx/2) + 4*(kqsx%2) + 2*j + l + 8] = val2;
#endif
            }
        }

        int ls = int(((bxi->scales_l[kqsx%4] >> 4*(kqsx/4)) & 0xf) | (((bxi->scales_h >> 2*kqsx) & 3) << 4)) - 32;

#ifdef INT8_MMA_AVAILABLE
        x_df[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx] = d * ls;
#else
        x_df[i*(WARP_SIZE/4) + i/4   + kqsx] = d * ls;
#endif
    }

}

template <int mmq_x, int mmq_y, bool need_check>
struct mmq_type_traits_id<mmq_x, mmq_y, need_check, GGML_TYPE_IQ2_KL> {
    static constexpr load_tiles_mmq_t load_tiles   = load_tiles_iq2_kl<mmq_y, need_check>;
    static constexpr vec_dot_mmq_t    vec_dot_mma  = vec_dot_q8_0_q8_1_mma<mmq_x, mmq_y, MMQ_Q8_1_DS_LAYOUT_D4>;
    static constexpr vec_dot_mmq_t    vec_dot_dp4a = vec_dot_q8_0_q8_1_dp4a<mmq_x, mmq_y>;
};

DECL_MMQ_CASE(GGML_TYPE_IQ2_KL);
