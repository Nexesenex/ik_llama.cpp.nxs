#include "hip/hip_runtime.h"
#include "../mmq_id_common.cuh"

template <int mmq_y, bool need_check> static __device__ __forceinline__ void load_tiles_iq2_ks(
    const char * __restrict__ x, int * __restrict__ x_tile, const int kbx0, const int i_max, const int stride) {

    constexpr int nwarps = mmq_get_nwarps_device();

#ifdef INT8_MMA_AVAILABLE
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + WARP_SIZE*2);
#else
    constexpr tile_x_sizes txs = mmq_get_dp4a_tile_x_sizes(GGML_TYPE_IQ4_XS, mmq_y);
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + txs.qs);
#endif // INT8_MMA_AVAILABLE

    const int kqsx = threadIdx.x%16;

#ifdef __CUDA_ARCH__
    #pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += 2*nwarps) {
        int i = i0 + 2*threadIdx.y + threadIdx.x/16;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_iq2_ks * bxi = (const block_iq2_ks *)(x + i*stride + sizeof(half)) + kbx0;

        uint16_t extra = bxi->extra >> 4*(kqsx/8);
        int q2 = get_int_b2(bxi->qs, kqsx);

        uint32_t extra32 = uint32_t(extra & 0xf) * 0x01010101;
        uint32_t val1 = ((q2 >> 0) & 0x33333333) | ((extra32 << 2) & 0x04040404) | ((extra32 << 4) & 0x40404040);
        uint32_t val2 = ((q2 >> 2) & 0x33333333) | ((extra32 << 1) & 0x04040404) | ((extra32 << 3) & 0x40404040);
        int2 v1 = get_int_from_table_8(val1, iq2nl_values);
        int2 v2 = get_int_from_table_8(val2, iq2nl_values);

#ifdef INT8_MMA_AVAILABLE
        x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx%8 + 32*(kqsx/8) +  0] = v1.x;
        x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx%8 + 32*(kqsx/8) +  8] = v2.x;
        x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx%8 + 32*(kqsx/8) + 16] = v1.y;
        x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx%8 + 32*(kqsx/8) + 24] = v2.y;
#else
        x_qs[i*(2*WARP_SIZE + 1)     + kqsx%8 + 32*(kqsx/8) +  0] = v1.x;
        x_qs[i*(2*WARP_SIZE + 1)     + kqsx%8 + 32*(kqsx/8) +  8] = v2.x;
        x_qs[i*(2*WARP_SIZE + 1)     + kqsx%8 + 32*(kqsx/8) + 16] = v1.y;
        x_qs[i*(2*WARP_SIZE + 1)     + kqsx%8 + 32*(kqsx/8) + 24] = v2.y;
#endif // INT8_MMA_AVAILABLE
    }

#else // __CUDA_ARCH__


    const int * all_values = (const int *)iq2k_table;
    #pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += 2*nwarps) {
        int i = i0 + 2*threadIdx.y + threadIdx.x/16;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_iq2_ks * bxi = (const block_iq2_ks *)(x + i*stride + sizeof(half)) + kbx0;

        uint16_t extra = bxi->extra >> 4*(kqsx/8);
        int q2 = get_int_b2(bxi->qs, kqsx);

#ifdef INT8_MMA_AVAILABLE
        x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx%8 + 32*(kqsx/8) +  0] = int_from_table_4((q2 >> 0) & 0x03030303, all_values + ((extra & 1) << 8));
        x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx%8 + 32*(kqsx/8) +  8] = int_from_table_4((q2 >> 2) & 0x03030303, all_values + ((extra & 2) << 7));
        x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx%8 + 32*(kqsx/8) + 16] = int_from_table_4((q2 >> 4) & 0x03030303, all_values + ((extra & 4) << 6));
        x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx%8 + 32*(kqsx/8) + 24] = int_from_table_4((q2 >> 6) & 0x03030303, all_values + ((extra & 8) << 5));
#else
        x_qs[i*(2*WARP_SIZE + 1)     + kqsx%8 + 32*(kqsx/8) +  0] = int_from_table_4((q2 >> 0) & 0x03030303, all_values + ((extra & 1) << 8));
        x_qs[i*(2*WARP_SIZE + 1)     + kqsx%8 + 32*(kqsx/8) +  8] = int_from_table_4((q2 >> 2) & 0x03030303, all_values + ((extra & 2) << 7));
        x_qs[i*(2*WARP_SIZE + 1)     + kqsx%8 + 32*(kqsx/8) + 16] = int_from_table_4((q2 >> 4) & 0x03030303, all_values + ((extra & 4) << 6));
        x_qs[i*(2*WARP_SIZE + 1)     + kqsx%8 + 32*(kqsx/8) + 24] = int_from_table_4((q2 >> 6) & 0x03030303, all_values + ((extra & 8) << 5));
#endif // INT8_MMA_AVAILABLE
    }
#endif // __CUDA_ARCH__

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * 8) {
        int i = i0 + threadIdx.y * 8 + threadIdx.x / 4;

        if (need_check) {
            i = min(i, i_max);
        }

        const half * dptr = (const half *)(x + i*stride);
        const float d = dptr[0];
        const block_iq2_ks * bxi = (const block_iq2_ks *)(dptr + 1) + kbx0;
        const int ls1 = ((bxi->scales[threadIdx.x % 4] >> 0) & 0xf) | ((bxi->extra >> (4 + 2*(threadIdx.x % 4))) & 0x10);
        const int ls2 = ((bxi->scales[threadIdx.x % 4] >> 4) & 0xf) | ((bxi->extra >> (5 + 2*(threadIdx.x % 4))) & 0x10);

#ifdef INT8_MMA_AVAILABLE
        x_df[i*MMQ_MMA_TILE_X_K_Q8_0 + 2*(threadIdx.x % 4) + 0] = d * (ls1 - 16);
        x_df[i*MMQ_MMA_TILE_X_K_Q8_0 + 2*(threadIdx.x % 4) + 1] = d * (ls2 - 16);
#else
        x_df[i*(WARP_SIZE/4) + i/4   + 2*(threadIdx.x % 4) + 0] = d * (ls1 - 16);
        x_df[i*(WARP_SIZE/4) + i/4   + 2*(threadIdx.x % 4) + 1] = d * (ls2 - 16);
#endif // INT8_MMA_AVAILABLE
    }
}

template <int mmq_x, int mmq_y, bool need_check>
struct mmq_type_traits_id<mmq_x, mmq_y, need_check, GGML_TYPE_IQ2_KS> {
    static constexpr load_tiles_mmq_t load_tiles   = load_tiles_iq2_ks<mmq_y, need_check>;
    static constexpr vec_dot_mmq_t    vec_dot_mma  = vec_dot_q8_0_q8_1_mma<mmq_x, mmq_y, MMQ_Q8_1_DS_LAYOUT_D4>;
    static constexpr vec_dot_mmq_t    vec_dot_dp4a = vec_dot_q8_0_q8_1_dp4a<mmq_x, mmq_y>;
};

DECL_MMQ_CASE(GGML_TYPE_IQ2_KS);
