#include "hip/hip_runtime.h"
#include "../mmq_id_common.cuh"

template <int mmq_y, bool need_check> static __device__ __forceinline__ void load_tiles_iq3_ks(
    const char * __restrict__ x, int * __restrict__ x_tile, const int kbx0, const int i_max, const int stride) {

    constexpr int nwarps = mmq_get_nwarps_device();

#ifdef INT8_MMA_AVAILABLE
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + WARP_SIZE*2);
#else
    constexpr tile_x_sizes txs = MMQ_DP4A_TXS_Q8_0_16;
    int   * x_qs = (int   *)  x_tile;
    float * x_df = (float *) (x_qs + txs.qs);
#endif // INT8_MMA_AVAILABLE

    constexpr int qstep = 8;
    const int kqsx = threadIdx.x % qstep;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * WARP_SIZE/qstep) {
        int i = i0 + threadIdx.y*(WARP_SIZE/qstep) + threadIdx.x/qstep;

        if (need_check) {
            i = min(i, i_max);
        }

        const half * dptr = (const half *)(x + i*stride);
        const float d = __half2float(dptr[0]);
        const block_iq3_ks * bxi = (const block_iq3_ks *)(dptr + 1) + kbx0;

        //uint16_t extra = bxi->extra >> 8;
        int qh = get_int_b2(bxi->qh, kqsx);

        uint32_t extra32 = uint32_t(bxi->extra >> 8) * 0x01010101;

        #pragma unroll
        for (int l = 0; l < qstep/4; ++l) {

            const int ql = get_int_b2(bxi->qs, kqsx + qstep*l);
            uint32_t val1 = ((ql >> 0) & 0x33333333) | ((qh << 2) & 0x04040404) | ((extra32 << 3) & 0x08080808)
                                                     | ((qh << 4) & 0x40404040) | ((extra32 << 5) & 0x80808080);
            uint32_t val2 = ((ql >> 2) & 0x33333333) | ((qh << 1) & 0x04040404) | ((extra32 << 2) & 0x08080808)
                                                     | ((qh << 3) & 0x40404040) | ((extra32 << 4) & 0x80808080);
            int2 v1 = get_int_from_table_16(val1, iq3nl_values);
            int2 v2 = get_int_from_table_16(val2, iq3nl_values);

            extra32 >>= 4;
            qh      >>= 4;

#ifdef INT8_MMA_AVAILABLE
            x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx + 32*l +  0] = v1.x;
            x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx + 32*l +  8] = v2.x;
            x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx + 32*l + 16] = v1.y;
            x_qs[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx + 32*l + 24] = v2.y;
#else
            x_qs[i*(2*WARP_SIZE + 1)     + kqsx + 32*l +  0] = v1.x;
            x_qs[i*(2*WARP_SIZE + 1)     + kqsx + 32*l +  8] = v2.x;
            x_qs[i*(2*WARP_SIZE + 1)     + kqsx + 32*l + 16] = v1.y;
            x_qs[i*(2*WARP_SIZE + 1)     + kqsx + 32*l + 24] = v2.y;
#endif // INT8_MMA_AVAILABLE
        }

#ifdef INT8_MMA_AVAILABLE
        x_df[i*MMQ_MMA_TILE_X_K_Q8_0 + kqsx] = d * (int(((bxi->scales[kqsx%4] >> 4*(kqsx/4)) & 0xf) | (((bxi->extra >> kqsx) & 1) << 4)) - 16);
#else
        x_df[i*(WARP_SIZE/4) + i/4   + kqsx] = d * (int(((bxi->scales[kqsx%4] >> 4*(kqsx/4)) & 0xf) | (((bxi->extra >> kqsx) & 1) << 4)) - 16);
#endif // INT8_MMA_AVAILABLE
    }
}

template <int mmq_x, int mmq_y, bool need_check>
struct mmq_type_traits_id<mmq_x, mmq_y, need_check, GGML_TYPE_IQ3_KS> {
    static constexpr load_tiles_mmq_t load_tiles   = load_tiles_iq3_ks<mmq_y, need_check>;
    static constexpr vec_dot_mmq_t    vec_dot_mma  = vec_dot_q8_0_q8_1_mma<mmq_x, mmq_y, MMQ_Q8_1_DS_LAYOUT_D4>;
    static constexpr vec_dot_mmq_t    vec_dot_dp4a = vec_dot_q8_0_q8_1_dp4a<mmq_x, mmq_y>;
};

DECL_MMQ_CASE(GGML_TYPE_IQ3_KS);
